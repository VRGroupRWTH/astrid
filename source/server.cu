#include <astrid/server.hpp>

#include <iostream>
#include <string>
#include <vector>

namespace ast
{
server::server(const std::int32_t port)
{
  if (communicator_.rank() == 0)
  {
    const auto address = std::string("tcp://*:") + std::to_string(port);
    socket_.bind(address);
    std::cout << "Socket bound at: " << address << ".\n";
  }
}

void server::run   ()
{
  proto::request            request     ;
  proto::image              response    ;
  std::int32_t              message_size;
  std::vector<std::uint8_t> message_data;
  image_type                image       ;

  while (!request.terminate())
  {
    if (communicator_.rank() == 0)
    {
      zmq::message_t message;
      socket_.recv(message, zmq::recv_flags::none);

      message_size = static_cast<std::int32_t>(message.size());
      message_data.resize(message.size());
      std::copy_n(static_cast<std::uint8_t*>(message.data()), message.size(), message_data.begin());
    }
    
#ifdef ASTRAY_USE_MPI
    communicator_.bcast (&message_size      , 1           , mpi::data_type(MPI_INT ));
    message_data .resize(message_size);
    communicator_.bcast (message_data.data(), message_size, mpi::data_type(MPI_BYTE));
#endif
    request.ParseFromArray(message_data.data(), static_cast<std::int32_t>(message_data.size()));
    
    update(request);
    std::visit([&] (auto& ray_tracer) { image = ray_tracer.render_frame(); }, ray_tracer_.value());

    if (communicator_.rank() == 0)
    {
      response.set_data(static_cast<void*>(image.data.data()), image.data.size() * sizeof(vector3<std::uint8_t>));
      response.mutable_size()->set_x (image.size[0]);
      response.mutable_size()->set_y (image.size[1]);
      auto string = response.SerializeAsString();

      zmq::message_t message(string.begin(), string.end());
      socket_.send(message, zmq::send_flags::none);
    }
  }
}

void server::update(const proto::request& request)
{
  if (request.has_metric())
  {
    ray_tracer_.reset();

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
      hipDeviceReset();
#endif

    ray_tracer_.emplace();

    if      (request.metric() == "alcubierre")
      ray_tracer_->emplace<ray_tracer<metrics::alcubierre                          <scalar_type>, motion_type>>();
    else if (request.metric() == "barriola_vilenkin")
      ray_tracer_->emplace<ray_tracer<metrics::barriola_vilenkin                   <scalar_type>, motion_type>>();
    else if (request.metric() == "bertotti_kasner")
      ray_tracer_->emplace<ray_tracer<metrics::bertotti_kasner                     <scalar_type>, motion_type>>();
    else if (request.metric() == "bessel")
      ray_tracer_->emplace<ray_tracer<metrics::bessel                              <scalar_type>, motion_type>>();
    else if (request.metric() == "de_sitter")
      ray_tracer_->emplace<ray_tracer<metrics::de_sitter                           <scalar_type>, motion_type>>();
    else if (request.metric() == "einstein_rosen_weber_wheeler_bonnor")
      ray_tracer_->emplace<ray_tracer<metrics::einstein_rosen_weber_wheeler_bonnor <scalar_type>, motion_type>>();
    else if (request.metric() == "friedman_lemaitre_robertson_walker")
      ray_tracer_->emplace<ray_tracer<metrics::friedman_lemaitre_robertson_walker  <scalar_type>, motion_type>>();
    else if (request.metric() == "goedel")
      ray_tracer_->emplace<ray_tracer<metrics::goedel                              <scalar_type>, motion_type>>();
    else if (request.metric() == "janis_newman_winicour")
      ray_tracer_->emplace<ray_tracer<metrics::janis_newman_winicour               <scalar_type>, motion_type>>();
    else if (request.metric() == "kastor_traschen")
      ray_tracer_->emplace<ray_tracer<metrics::kastor_traschen                     <scalar_type>, motion_type>>();
    else if (request.metric() == "kerr")
      ray_tracer_->emplace<ray_tracer<metrics::kerr                                <scalar_type>, motion_type>>();
    else if (request.metric() == "kottler")
      ray_tracer_->emplace<ray_tracer<metrics::kottler                             <scalar_type>, motion_type>>();
    else if (request.metric() == "minkowski")
      ray_tracer_->emplace<ray_tracer<metrics::minkowski                           <scalar_type>, motion_type>>();
    else if (request.metric() == "morris_thorne")
      ray_tracer_->emplace<ray_tracer<metrics::morris_thorne                       <scalar_type>, motion_type>>();
    else if (request.metric() == "reissner_nordstroem")
      ray_tracer_->emplace<ray_tracer<metrics::reissner_nordstroem                 <scalar_type>, motion_type>>();
    else if (request.metric() == "reissner_nordstroem_extreme_dihole")
      ray_tracer_->emplace<ray_tracer<metrics::reissner_nordstroem_extreme_dihole  <scalar_type>, motion_type>>();
    else if (request.metric() == "schwarzschild")
      ray_tracer_->emplace<ray_tracer<metrics::schwarzschild                       <scalar_type>, motion_type>>();
    else if (request.metric() == "schwarzschild_cosmic_string")
      ray_tracer_->emplace<ray_tracer<metrics::schwarzschild_cosmic_string         <scalar_type>, motion_type>>();
  }

  std::visit([&] (auto& ray_tracer)
  {
    if (request.has_image_size       ())
      ray_tracer.set_image_size      ({request.image_size().x(), request.image_size().y()});
    if (request.has_iterations       ())
      ray_tracer.set_iterations      (request.iterations());
    if (request.has_lambda_step_size ())
      ray_tracer.set_lambda_step_size(request.lambda_step_size());
    if (request.has_lambda           ())
      ray_tracer.set_lambda          (request.lambda());
    if (request.has_debug            ())
      ray_tracer.set_debug           (request.debug());
    
    if (request.has_bounds          ())
    {
      auto& bounds = request.bounds();
      auto& lower  = bounds .lower ();
      auto& upper  = bounds .upper ();
      ray_tracer.set_bounds(aabb4<scalar_type>(
        vector4<scalar_type>(lower.t(), lower.x(), lower.y(), lower.z()),
        vector4<scalar_type>(upper.t(), upper.x(), upper.y(), upper.z())));
    }

    if (request.has_transform       ())
    {
      auto& transform = request.transform();

      if (transform.has_time          ())
        ray_tracer.observer().set_coordinate_time(request.transform().time());

      if (transform.has_position      ())
      {
        auto& position = transform.position();
        ray_tracer.observer().transform().translation = {position.x(), position.y(), position.z()};
      }
      
      if (transform.has_rotation_euler())
      {
        auto& rotation = transform.rotation_euler();
        ray_tracer.observer().transform().rotation_from_euler({to_radians(rotation.x()), to_radians(rotation.y()), to_radians(rotation.z())});
      }

      if (transform.has_look_at_origin() && transform.look_at_origin())
        ray_tracer.observer().transform().look_at({0, 0, 0});
    }

    if (request.has_projection      ())
    {
      const auto& projection   = request.projection();
      const auto& image_size   = ray_tracer.image_size();
      const auto  aspect_ratio = static_cast<scalar_type>(image_size[0]) / static_cast<scalar_type>(image_size[1]);

      if (projection.has_type())
      {
        if      (projection.type() == "perspective" )
          ray_tracer.observer().set_projection(perspective_projection <scalar_type>{to_radians<scalar_type>(75), aspect_ratio});
        else if (projection.type() == "orthographic")
          ray_tracer.observer().set_projection(orthographic_projection<scalar_type>{1, aspect_ratio});
      }

      if      (std::holds_alternative<perspective_projection <scalar_type>>(ray_tracer.observer().projection()))
      {
        auto& cast_projection = std::get<perspective_projection<scalar_type>>(ray_tracer.observer().projection());

        if (request.has_image_size())
          cast_projection.aspect_ratio = aspect_ratio;
        if (projection.has_y_field_of_view())
          cast_projection.fov_y        = to_radians(projection.y_field_of_view());
        if (projection.has_focal_length   ())
          cast_projection.focal_length = projection.focal_length   ();
        if (projection.has_near_clip      ())
          cast_projection.near_clip    = projection.near_clip      ();
        if (projection.has_far_clip       ())
          cast_projection.far_clip     = projection.far_clip       ();
      }
      else if (std::holds_alternative<orthographic_projection<scalar_type>>(ray_tracer.observer().projection()))
      {
        auto& cast_projection = std::get<orthographic_projection<scalar_type>>(ray_tracer.observer().projection());
      
        if (request.has_image_size())
          cast_projection.aspect_ratio = aspect_ratio;
        if (projection.has_height         ())
          cast_projection.height       = projection.height         ();
        if (projection.has_near_clip      ())
          cast_projection.near_clip    = projection.near_clip      ();
        if (projection.has_far_clip       ())
          cast_projection.far_clip     = projection.far_clip       ();
      }
    }

    if (request.has_background_image())
    {
      auto& background = request.background_image();

      image_type image(vector2<std::int32_t>{background.size().x(), background.size().y()});
      std::copy_n(background.data().data(), background.data().size(), reinterpret_cast<std::uint8_t*>(image.data.data()));
      ray_tracer.set_background(image);
    }
  }, ray_tracer_.value());
}
}