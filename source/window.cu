#include "hip/hip_runtime.h"
#include <astrid/window.hpp>

#include <QFileDialog>
#include <QInputDialog>
#include <QKeyEvent>
#include <QLineEdit>
#include <QString>

namespace ast
{
window::window(QWidget* parent) : QMainWindow(parent), ui_(new Ui::main_window)
{
  ui_->setupUi(this);
  
  resize(1024, 512);
  
  connect(ui_->action_connect_local          , &QAction    ::triggered         , this, [&] 
  {
    create_client();
  });
  connect(ui_->action_connect_remote         , &QAction    ::triggered         , this, [&] 
  {
    bool confirm;
    const auto address = QInputDialog::getText(
      this, 
      "Connect", 
      "Enter the IP address and port of the Astrid server:",
      QLineEdit::Normal,
      "127.0.0.1:3000",
      &confirm);

    if (confirm)
      create_client(address.toStdString());
  });
  connect(ui_->action_disconnect             , &QAction    ::triggered         , this, [&] 
  {
    destroy_client();
  });
  connect(ui_->action_exit                   , &QAction    ::triggered         , this, [&] 
  {
    std::exit(0);
  });
  
  connect(ui_->button_iterations_05          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_iterations;
    const auto value     = line_edit->text().toULongLong() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_iterations_2           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_iterations;
    const auto value     = line_edit->text().toULongLong() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_step_size_05    , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda_step_size;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_step_size_2     , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda_step_size;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_minus_1         , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda;
    const auto value     = line_edit->text().toFloat() - 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_plus_1          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda;
    const auto value     = line_edit->text().toFloat() + 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_coordinate_time_minus_1, &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_coordinate_time;
    const auto value     = line_edit->text().toFloat() - 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_coordinate_time_plus_1 , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_coordinate_time;
    const auto value     = line_edit->text().toFloat() + 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_fov_y_05               , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_fov_y;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_fov_y_2                , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_fov_y;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_focal_length_05        , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_focal_length;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_focal_length_2         , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_focal_length;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_size_ortho_05          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_size_ortho;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_size_ortho_2           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_size_ortho;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_near_clip_05           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_near_clip;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_near_clip_2            , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_near_clip;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_far_clip_05            , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_far_clip;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_far_clip_2             , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_far_clip;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_background_browse      , &QPushButton::clicked           , this, [&]
  {
    const QString filepath = QFileDialog::getOpenFileName(
      this, 
      "Select background image file.",
      QString(),
      "Images (*.bmp *.jpg *.png *.tga)");

    if (!filepath.isNull())
    {
      const auto line_edit = ui_->line_edit_background;
      line_edit->setText(filepath);
      background_.load(line_edit->text().toStdString());
    }
  });
  connect(ui_->button_render                 , &QPushButton::clicked           , this, [&]
  {
    if (client_)
    {
      client_->make_request();
      ui_->button_render->setEnabled(false);
    }
  });
  
  connect(ui_->checkbox_autorender           , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    if (client_)
      client_->set_auto_request(ui_->checkbox_autorender->isChecked());

    ui_->button_render->setEnabled(!checked);
  });
  connect(ui_->checkbox_use_bounds           , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    ui_->line_edit_lower_bound_t->setEnabled(checked);
    ui_->line_edit_lower_bound_x->setEnabled(checked);
    ui_->line_edit_lower_bound_y->setEnabled(checked);
    ui_->line_edit_lower_bound_z->setEnabled(checked);
    ui_->line_edit_upper_bound_t->setEnabled(checked);
    ui_->line_edit_upper_bound_x->setEnabled(checked);
    ui_->line_edit_upper_bound_y->setEnabled(checked);
    ui_->line_edit_upper_bound_z->setEnabled(checked);
  });
  connect(ui_->checkbox_look_at_origin       , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    ui_->line_edit_rotation_x   ->setEnabled(!checked);
    ui_->line_edit_rotation_y   ->setEnabled(!checked);
    ui_->line_edit_rotation_z   ->setEnabled(!checked);
  });
  
  connect(ui_->combobox_projection_type      , &QComboBox  ::currentTextChanged, this, [&] (const QString& text)
  {
    const bool is_perspective = text == "perspective";
    ui_->line_edit_fov_y       ->setEnabled( is_perspective);
    ui_->button_fov_y_05       ->setEnabled( is_perspective);
    ui_->button_fov_y_2        ->setEnabled( is_perspective);
    ui_->line_edit_focal_length->setEnabled( is_perspective);
    ui_->button_focal_length_05->setEnabled( is_perspective);
    ui_->button_focal_length_2 ->setEnabled( is_perspective);
    ui_->line_edit_size_ortho  ->setEnabled(!is_perspective);
    ui_->button_size_ortho_05  ->setEnabled(!is_perspective);
    ui_->button_size_ortho_2   ->setEnabled(!is_perspective);
  });
   
  statusBar()->showMessage("Initialization successful.");
}

void window::set_ui_state     (const bool connected) const
{
  ui_->action_connect_local ->setEnabled(!connected);
  ui_->action_connect_remote->setEnabled(!connected);
  ui_->action_disconnect    ->setEnabled( connected);
  ui_->frame_render         ->setEnabled( connected);
  ui_->toolbox              ->setEnabled( connected);
}

void window::create_client    (const std::string& address)
{
  statusBar()->showMessage("Connecting to " + QString::fromStdString(address) + ". Please wait.");
  repaint  ();

  try
  {
    client_ = std::make_unique<client>(
      [&] (proto::request& request_data)
      {
        statusBar()->showMessage("Sending render request to the server. Please wait.");
        
        fill_request_data(request_data);
      },
      [&] (const proto::image& response_data)
      {
        statusBar()->showMessage("Received image from the server.");

        ui_->image->setPixmap(QPixmap::fromImage(QImage(
          reinterpret_cast<const unsigned char*>(response_data.data().data()),
          response_data.size().x(),
          response_data.size().y(),
          response_data.size().x() * sizeof(vector3<std::uint8_t>),
          QImage::Format_RGB888)));

        if(!ui_->checkbox_autorender->isChecked())
          ui_->button_render->setEnabled(true);
      },
      [&]
      {
        statusBar()->showMessage("Disconnected from the server.");
        set_ui_state(false);
        client_.reset();
      }
      ,address);
    
    statusBar()->showMessage("Connected to " + QString::fromStdString(address) + ".");
    set_ui_state(true);
  }
  catch (const std::runtime_error&)
  {
    statusBar()->showMessage("Failed to connect " + QString::fromStdString(address) + ".");
  }
}
void window::destroy_client   () 
{
  statusBar()->showMessage("Disconnecting from the server. Please wait.");
  client_->kill();
}

void window::fill_request_data(proto::request& request)
{
  // Always set parameters.
  {
    request.mutable_image_size       ()->set_x(ui_->image->width () - 2 * ui_->image->frameWidth  ());
    request.mutable_image_size       ()->set_y(ui_->image->height() - 2 * ui_->image->frameWidth  ());
    request.set_iterations           (         ui_->line_edit_iterations      ->text().toULongLong());
    request.set_lambda_step_size     (         ui_->line_edit_lambda_step_size->text().toFloat    ());
    request.set_lambda               (         ui_->line_edit_lambda          ->text().toFloat    ());
    request.set_debug                (         ui_->checkbox_debug            ->isChecked         ());
    
    if (ui_->checkbox_use_bounds->isChecked())
    {
      request.mutable_bounds()->mutable_lower()->set_t(ui_->line_edit_lower_bound_t->text().toFloat());
      request.mutable_bounds()->mutable_lower()->set_x(ui_->line_edit_lower_bound_x->text().toFloat());
      request.mutable_bounds()->mutable_lower()->set_y(ui_->line_edit_lower_bound_y->text().toFloat());
      request.mutable_bounds()->mutable_lower()->set_z(ui_->line_edit_lower_bound_z->text().toFloat());
      request.mutable_bounds()->mutable_upper()->set_t(ui_->line_edit_upper_bound_t->text().toFloat());
      request.mutable_bounds()->mutable_upper()->set_x(ui_->line_edit_upper_bound_x->text().toFloat());
      request.mutable_bounds()->mutable_upper()->set_y(ui_->line_edit_upper_bound_y->text().toFloat());
      request.mutable_bounds()->mutable_upper()->set_z(ui_->line_edit_upper_bound_z->text().toFloat());
    }
    else
    {
      aabb4<float> empty_aabb;
      request.mutable_bounds()->mutable_lower()->set_t(empty_aabb.min()[0]);
      request.mutable_bounds()->mutable_lower()->set_x(empty_aabb.min()[1]);
      request.mutable_bounds()->mutable_lower()->set_y(empty_aabb.min()[2]);
      request.mutable_bounds()->mutable_lower()->set_z(empty_aabb.min()[3]);
      request.mutable_bounds()->mutable_upper()->set_t(empty_aabb.max()[0]);
      request.mutable_bounds()->mutable_upper()->set_x(empty_aabb.max()[1]);
      request.mutable_bounds()->mutable_upper()->set_y(empty_aabb.max()[2]);
      request.mutable_bounds()->mutable_upper()->set_z(empty_aabb.max()[3]);
    }

    const auto transform = request.mutable_transform();
    transform->set_time              (         ui_->line_edit_coordinate_time ->text().toFloat    ());
    transform->mutable_position      ()->set_x(ui_->line_edit_position_x      ->text().toFloat    ());
    transform->mutable_position      ()->set_y(ui_->line_edit_position_y      ->text().toFloat    ());
    transform->mutable_position      ()->set_z(ui_->line_edit_position_z      ->text().toFloat    ());
    transform->mutable_rotation_euler()->set_x(ui_->line_edit_rotation_x      ->text().toFloat    ());
    transform->mutable_rotation_euler()->set_y(ui_->line_edit_rotation_y      ->text().toFloat    ());
    transform->mutable_rotation_euler()->set_z(ui_->line_edit_rotation_z      ->text().toFloat    ());
    transform->set_look_at_origin    (         ui_->checkbox_look_at_origin   ->isChecked         ());

    request.mutable_projection()->set_type(ui_->combobox_projection_type->currentText().toStdString());

    if      (ui_->combobox_projection_type->currentText() == "perspective" )
    {
      request.mutable_projection()->set_y_field_of_view(ui_->line_edit_fov_y       ->text().toFloat());
      request.mutable_projection()->set_focal_length   (ui_->line_edit_focal_length->text().toFloat());
      request.mutable_projection()->set_near_clip      (ui_->line_edit_near_clip   ->text().toFloat());
      request.mutable_projection()->set_far_clip       (ui_->line_edit_far_clip    ->text().toFloat());
    }
    else if (ui_->combobox_projection_type->currentText() == "orthographic")
    {
      request.mutable_projection()->set_type           ("orthographic");
      request.mutable_projection()->set_height         (ui_->line_edit_size_ortho  ->text().toFloat());
      request.mutable_projection()->set_near_clip      (ui_->line_edit_near_clip   ->text().toFloat());
      request.mutable_projection()->set_far_clip       (ui_->line_edit_far_clip    ->text().toFloat());
    }
  }

  // Conditional parameters.
  static QString cached_background, cached_metric;

  if (!ui_->line_edit_background->text       ().isNull () &&
      !ui_->line_edit_background->text       ().isEmpty() &&
       ui_->combobox_metric     ->currentText() != cached_metric ||
       ui_->line_edit_background->text       () != cached_background)
  {
    const auto image = request.mutable_background_image();
    image->set_data(static_cast<void*>(background_.data.data()), background_.data.size() * sizeof(vector3<std::uint8_t>));
    image->mutable_size()->set_x(background_.size[0]);
    image->mutable_size()->set_y(background_.size[1]);
    cached_background = ui_->line_edit_background->text();
  }
  else
    request.clear_background_image();
    
  if (ui_->combobox_metric->currentText() != cached_metric)
  {
    const auto text = ui_->combobox_metric->currentText();
    *request.mutable_metric() = text.toStdString();
    cached_metric = text;
  }
  else
    request.clear_metric();
}
}